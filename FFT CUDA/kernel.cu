#include "hip/hip_runtime.h"
/*
Author: Sharome Burton
Date: 20/June/2020
Title: CUDA Parallel Implementation of Radix 2 FFT

This program runs from kernel.cu.
This program executes the Radix-2 Fast Fourier Transform on two 32768-element arrays
multiple times using GPU parallel processing capabilities. The time taken to execute the main function is measured using the clock()
function
*/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>


hipError_t CudaFFT(const float *RX, const float *IX, float *RFFT, float *IFFT, unsigned int size);

__global__ void FFT(const float *RX, const float *IX, float *RFFT, float *IFFT, int size)
{
	const float PI = 3.14159;
	float numElements = 4;
	int i = numElements * (blockDim.x * blockIdx.x +  threadIdx.x);

	if (i < size) 
	{
		for (int num = 0; num < numElements; num += 2)
		{
		// Even - 2n eg. {0,2,4,6, etc.}
		
			// Real
			RFFT[i + num] = (RX[i + num] * (cosf(2 * PI*(i + num) / size))) + (IX[i + num] * 1 * (sinf(2 * PI * (i + num) / size)));
			// Imaginary
			IFFT[i + num] = (RX[i + num] * -1 * (sinf(2 * PI*(i + num) / size))) + (IX[i + num] * (cosf(2 * PI * (i + num) / size)));


		// Odd - 2n-1 eg. {1,3,5,7, etc.}

				// Real
			RFFT[i + num + 1] = (RX[i + num + 1] * (cosf(2 * PI*(i + num + 1) / size))) + (IX[i + num + 1] * 1 * (sinf(2 * PI * (i + num + 1) / size)));
			// Imaginary
			IFFT[i + num + 1] = (RX[i + num + 1] * -1 * (sinf(2 * PI*(i + num + 1) / size))) + (IX[i + num + 1] * (cosf(2 * PI * (i + num + 1) / size)));
		}
	}
}

const int ARRAYSIZE = 32768;

int main()
{

	// Time Check
	clock_t t;
	t = clock();
	srand(time(NULL));

	// FFT declarations
	float RX[ARRAYSIZE];	// Real X values
	float IX[ARRAYSIZE];	// Imaginary X values
	float RFFT[ARRAYSIZE];	// Real FFT components
	float IFFT[ARRAYSIZE];	// Imaginary FFT components
	float RSUM = 0;	// Sum of real components
	float ISUM = 0; // Sum of imaginary components


	// Filling input arrays with numbers from 0.0-10.0
	for (int i = 0; i < ARRAYSIZE; i++)
	{
		RX[i] = (rand() % 100) / 10.0;
		IX[i] = (rand() % 100) / 10.0;
	}

	// FFT Process

	hipError_t cudaStatus = CudaFFT(RX, IX, RFFT, IFFT, ARRAYSIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CudaFFT failed!");
		return 1;
	}

	for (int i = 0; i < 9999; i++)
	{
		CudaFFT(RX, IX, RFFT, IFFT, ARRAYSIZE);
	}

	// FFT Result

	for (int i = 0; i < ARRAYSIZE; i++)
	{
		RSUM += RFFT[i];
		ISUM += IFFT[i];
	}

	printf("The sum of real components of X is: %f \n The sum of imaginary components of X is: %f \n", RSUM, ISUM);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	// Shows time elapsed
	t = clock() - t;
	double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds
	printf("FFT of 32768-element arrays 9999 times took %f seconds to execute using CUDA\n", time_taken);

	return 0;
}


// Helper function for using CUDA to do FFT in parallel 
hipError_t CudaFFT(const float *RX, const float *IX, float *RFFT, float *IFFT, unsigned int size)
{
	float *dev_RX = 0;
	float *dev_IX = 0;
	float *dev_RFFT = 0;
	float *dev_IFFT = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for four vectors (two input, two output)
	cudaStatus = hipMalloc((void**)&dev_RX, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_IX, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_RFFT, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_IFFT, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_RX, RX, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_IX, IX, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	FFT<<< size/128, 128/4 >>>(dev_RX, dev_IX, dev_RFFT, dev_IFFT, ARRAYSIZE);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "FFT launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(RFFT, dev_RFFT, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(IFFT, dev_IFFT, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_RX);
	hipFree(dev_IX);
	hipFree(dev_RFFT);
	hipFree(dev_IFFT);

	return cudaStatus;
}